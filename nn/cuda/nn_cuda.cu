#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../structs.h"
#include "../activation.h"
extern "C" void fc_forward_CUDA(FCLayer *layer, float *input);

__global__ void matrix_mul(float *input, float *weights, float *biases, float *output, int input_size, int output_size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < output_size){
        float result = biases[i];

        for (int j = 0; j < input_size; ++j){
            result += input[j] * weights[j  * output_size + i];
        }

        output[i] = result;
    }
}

extern "C" void fc_forward_CUDA(FCLayer *layer, float *input){
    // define pointers to the gpu
    float *gpu_input, *gpu_weights, *gpu_biases, *gpu_output;

    // allocate memory for the gpu
    hipMalloc((float**)&gpu_input, layer->input_size * sizeof(float));
    hipMalloc((float**)&gpu_weights, layer->input_size * layer->output_size * sizeof(float));
    hipMalloc((float**)&gpu_biases, layer->output_size * sizeof(float));
    hipMalloc((float**)&gpu_output, layer->output_size * sizeof(float));

    // Copy data from host (CPU) to device (GPU)
    hipMemcpy(gpu_input, input, layer->input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_weights, layer->weights, layer->input_size * layer->output_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_biases, layer->biases, layer->output_size * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int blockSize = 256;
    int gridSize = (layer->output_size + blockSize - 1) / blockSize;

    // Launch the CUDA kernel
    matrix_mul<<<gridSize, blockSize>>>(gpu_input, gpu_weights, gpu_biases, gpu_output, layer->input_size, layer->output_size);

    hipDeviceSynchronize();

    // Copy the output data back from the GPU to the CPU
    hipMemcpy(layer->output, gpu_output, layer->output_size * sizeof(float), hipMemcpyDeviceToHost);
    

    // Free the GPU memory
    hipFree(gpu_input);
    hipFree(gpu_weights);
    hipFree(gpu_biases);
    hipFree(gpu_output);

    // // get the activation function
    // float (*acti_func)(float,bool) = apply_activation(layer->activation_function);

    // // apply activation function on outputs
    // for (int i = 0; i < layer->output_size; ++i){
    //     float a = acti_func(layer->output[i], false);
    //     layer->output[i] = a;
    // }
}